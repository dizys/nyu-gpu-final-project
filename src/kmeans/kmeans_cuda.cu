#include <iostream>
#include <fstream>
#include <cfloat>
#include <cstdlib>
#include <time.h>
#include <ctime>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_NUM 8
#define BLOCK_SIZE 500

#define K 10
#define DIM 3

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

float *parse_input(const std::string &filename, long unsigned &vector_size)
{
    std::ifstream input;
    input.open(filename.c_str());
    if (!input.is_open())
    {
        std::cout << "Error: cannot open file at \"" << filename << "\"" << std::endl;
        exit(1);
    }
    if (!(input >> vector_size))
    {
        std::cout << "Error: cannot read vector size" << std::endl;
        exit(1);
    }

    float *vector = (float *)malloc(vector_size * DIM * sizeof(float));
    for (long unsigned i = 0; i < vector_size * DIM; i++)
    {
        if (!(input >> vector[i]))
        {
            std::cout << "Error: cannot read vector element" << std::endl;
            exit(1);
        }
    }

    input.close();
    return vector;
}

void pick_random_centroids(float *centroids, float *vector, long unsigned vector_size)
{
    for (int i = 0; i < K; i++)
    {
        int centroid_index = rand() % vector_size;
        for (int j = 0; j < DIM; j++)
        {
            centroids[i * DIM + j] = vector[centroid_index * DIM + j];
        }
    }
}

// kernel: reassigns each vector to the closest centroid
__global__ void kernel_cluster(unsigned vector_size, unsigned vector_stride, float *vectors, float *centroids, unsigned *clusters, unsigned *cluster_sizes, bool *changed)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i == 0)
    {
        changed[0] = false;

        for (int i = 0; i < K; i++)
        {
            cluster_sizes[i] = 0;
        }
    }

    __syncthreads();

    for (unsigned j = i * vector_stride; j < (i + 1) * vector_stride && j < vector_size; j++)
    {
        float min_dist = FLT_MAX;
        unsigned min_centroid = 0;
        for (unsigned k = 0; k < K; k++)
        {
            float dist = 0;
            for (unsigned l = 0; l < DIM; l++)
            {
                float diff = vectors[j * DIM + l] - centroids[k * DIM + l];
                dist += diff * diff;
            }
            if (dist < min_dist)
            {
                min_dist = dist;
                min_centroid = k;
            }
        }

        if (clusters[j] != min_centroid)
        {
            clusters[j] = min_centroid;
            changed[0] = true;
        }

        atomicAdd(&cluster_sizes[min_centroid], 1);
    }
}

// kernel: computes the new centroids
__global__ void kernel_compute_centroids(unsigned vector_size, unsigned vector_stride, float *vectors, float *centroids, unsigned *clusters, unsigned *cluster_sizes)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i == 0)
    {
        for (unsigned j = 0; j < K; j++)
        {
            for (unsigned k = 0; k < DIM; k++)
            {
                centroids[j * DIM + k] = 0;
            }
        }
    }

    __syncthreads();

    for (unsigned j = i * vector_stride; j < (i + 1) * vector_stride && j < vector_size; j++)
    {
        unsigned cluster = clusters[j];
        for (unsigned k = 0; k < DIM; k++)
        {
            atomicAdd(&centroids[cluster * DIM + k], vectors[j * DIM + k]);
        }
    }

    __syncthreads();

    if (i == 0)
    {
        for (unsigned j = 0; j < K; j++)
        {
            for (unsigned k = 0; k < DIM; k++)
            {
                if (cluster_sizes[j] > 0)
                {
                    centroids[j * DIM + k] /= cluster_sizes[j];
                }
            }
        }
    }
}

int main(int argc, char *argv[])
{
    std::srand(std::time(nullptr));
    if (argc != 2)
    {
        std::cout << "usage: " << argv[0] << " filename" << std::endl;
        return 1;
    }
    std::string filename = argv[1];
    long unsigned vector_size = 0;
    float *vectors = parse_input(filename, vector_size);
    float *centroids = (float *)malloc(K * DIM * sizeof(float));
    unsigned *clusters = (unsigned *)malloc(vector_size * sizeof(unsigned));
    for (unsigned i = 0; i < vector_size; i++)
    {
        clusters[i] = 0;
    }
    pick_random_centroids(centroids, vectors, vector_size);

    struct timespec start_time, end_time;

    clock_gettime(CLOCK_REALTIME, &start_time);

    float *d_vectors, *d_centroids;
    unsigned *d_clusters, *d_cluster_sizes;

    hipMalloc((void **)&d_vectors, vector_size * DIM * sizeof(float));
    hipMalloc((void **)&d_centroids, K * DIM * sizeof(float));
    hipMalloc((void **)&d_clusters, vector_size * sizeof(unsigned));
    hipMalloc((void **)&d_cluster_sizes, K * sizeof(unsigned));

    dim3 grid_size(BLOCK_NUM, 1, 1);
    dim3 block_size(BLOCK_SIZE, 1, 1);
    unsigned vector_stride = ceil(vector_size / (float)(grid_size.x * block_size.x));

    hipMemcpy(d_vectors, vectors, vector_size * DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids, K * DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_clusters, clusters, vector_size * sizeof(unsigned), hipMemcpyHostToDevice);

    bool *changed = (bool *)malloc(sizeof(bool));
    changed[0] = true;
    bool *d_changed;
    hipMalloc((void **)&d_changed, sizeof(bool));

    int iteration = 0;
    while (changed[0])
    {
        kernel_cluster<<<grid_size, block_size>>>(vector_size, vector_stride, d_vectors, d_centroids, d_clusters, d_cluster_sizes, d_changed);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        kernel_compute_centroids<<<grid_size, block_size>>>(vector_size, vector_stride, d_vectors, d_centroids, d_clusters, d_cluster_sizes);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipMemcpy(changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
        iteration++;
        std::cout << "Iteration #" << iteration << ": " << (changed[0] ? "centroids changed, continuing..." : "converged.") << std::endl;
    }

    hipMemcpy(clusters, d_clusters, vector_size * sizeof(unsigned), hipMemcpyDeviceToHost);

    hipFree(d_vectors);
    hipFree(d_centroids);
    hipFree(d_clusters);
    hipFree(d_cluster_sizes);
    hipFree(d_changed);

    clock_gettime(CLOCK_REALTIME, &end_time);

    printf("Total time taken by the GPU part = %lf\n", (double)(end_time.tv_sec - start_time.tv_sec) + (double)(end_time.tv_nsec - start_time.tv_nsec) / 1000000000);

    free(changed);
    free(vectors);
    free(centroids);
    free(clusters);

    return 0;
}
