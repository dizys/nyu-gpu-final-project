#include <iostream>
#include <fstream>
#include <cfloat>
#include <cstdlib>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_NUM 8
#define BLOCK_SIZE 500

#define K 10
#define DIM 3

float *parse_input(const std::string &filename, long unsigned &vector_size)
{
    std::ifstream input;
    input.open(filename.c_str());
    if (!input.is_open())
    {
        std::cout << "Error: cannot open file at \"" << filename << "\"" << std::endl;
        exit(1);
    }
    if (!(input >> vector_size))
    {
        std::cout << "Error: cannot read vector size" << std::endl;
        exit(1);
    }

    float *vector = (float *)malloc(vector_size * DIM * sizeof(float));
    for (long unsigned i = 0; i < vector_size * DIM; i++)
    {
        if (!(input >> vector[i]))
        {
            std::cout << "Error: cannot read vector element" << std::endl;
            exit(1);
        }
    }

    input.close();
    return vector;
}

void pick_random_centroids(float *centroids, float *vector, long unsigned vector_size)
{
    for (int i = 0; i < K; i++)
    {
        int centroid_index = rand() % vector_size;
        for (int j = 0; j < DIM; j++)
        {
            centroids[i * DIM + j] = vector[centroid_index * DIM + j];
        }
    }
}

// kernel: reassigns each vector to the closest centroid + computes the new centroids
__global__ void kernel(unsigned vector_size, unsigned vector_stride, float *vectors, float *centroids, unsigned *clusters, unsigned *cluster_sizes, bool *changed)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i == 0)
    {
        changed[0] = false;

        for (int i = 0; i < K; i++)
        {
            cluster_sizes[i] = 0;
        }

        printf("printing from kernel\n");
    }

    __syncthreads();

    for (unsigned j = i * vector_stride; j < (i + 1) * vector_stride && j < vector_size; j++)
    {
        float min_dist = FLT_MAX;
        unsigned min_centroid = 0;
        for (unsigned k = 0; k < K; k++)
        {
            float dist = 0;
            for (unsigned l = 0; l < DIM; l++)
            {
                float diff = vectors[j * DIM + l] - centroids[k * DIM + l];
                dist += diff * diff;
            }
            if (dist < min_dist)
            {
                min_dist = dist;
                min_centroid = k;
            }
        }

        if (clusters[j] != min_centroid)
        {
            clusters[j] = min_centroid;
            changed[0] = true;
        }
        atomicAdd(&cluster_sizes[min_centroid], 1);
    }

    __syncthreads();

    if (i == 0)
    {
        for (unsigned j = 0; j < K; j++)
        {
            for (unsigned k = 0; k < DIM; k++)
            {
                centroids[j * DIM + k] = 0;
            }
        }
    }

    __syncthreads();

    for (unsigned j = i * vector_stride; j < (i + 1) * vector_stride && j < vector_size; j++)
    {
        unsigned cluster = clusters[j];
        for (unsigned k = 0; k < DIM; k++)
        {
            atomicAdd(&centroids[cluster * DIM + k], vectors[j * DIM + k]);
        }
    }

    __syncthreads();

    if (i == 0)
    {
        for (unsigned j = 0; j < K; j++)
        {
            for (unsigned k = 0; k < DIM; k++)
            {
                if (cluster_sizes[j] > 0)
                {
                    centroids[j * DIM + k] /= cluster_sizes[j];
                }
            }
        }
    }
}

int main(int argc, char *argv[])
{
    if (argc != 2)
    {
        std::cout << "usage: " << argv[0] << " filename" << std::endl;
        return 1;
    }
    std::string filename = argv[1];
    long unsigned vector_size = 0;
    float *vectors = parse_input(filename, vector_size);
    float *centroids = (float *)malloc(K * DIM * sizeof(float));
    unsigned *clusters = (unsigned *)malloc(vector_size * sizeof(unsigned));
    for (unsigned i = 0; i < vector_size; i++)
    {
        clusters[i] = 1;
    }
    pick_random_centroids(centroids, vectors, vector_size);
    for (int i = 0; i < K; i++)
    {
        for (int j = 0; j < DIM; j++)
        {
            std::cout << centroids[i * DIM + j] << " ";
        }
        std::cout << std::endl;
    }

    struct timespec start_time, end_time;

    clock_gettime(CLOCK_REALTIME, &start_time);

    float *d_vectors, *d_centroids;
    unsigned *d_clusters, *d_cluster_sizes;

    hipMalloc((void **)&d_vectors, vector_size * DIM * sizeof(float));
    hipMalloc((void **)&d_centroids, K * DIM * sizeof(float));
    hipMalloc((void **)&d_clusters, vector_size * sizeof(unsigned));
    hipMalloc((void **)&d_cluster_sizes, K * sizeof(unsigned));

    dim3 grid_size(BLOCK_NUM, 1, 1);
    dim3 block_size(BLOCK_SIZE, 1, 1);
    unsigned vector_stride = ceil(vector_size / (float)(grid_size.x * block_size.x));

    hipMemcpy(d_vectors, vectors, vector_size * DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids, K * DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_clusters, clusters, vector_size * sizeof(unsigned), hipMemcpyHostToDevice);

    bool *changed = (bool *)malloc(sizeof(bool));
    changed[0] = true;
    bool *d_changed;
    hipMalloc((void **)&d_changed, sizeof(bool));
    int iteration = 0;
    std::cout << "stride: " << vector_stride << std::endl;
    while (changed[0] && iteration < 100)
    {
        kernel<<<grid_size, block_size>>>(vector_size, vector_stride, d_vectors, d_centroids, d_clusters, d_cluster_sizes, d_changed);
        hipMemcpy(changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(clusters, d_clusters, vector_size * sizeof(unsigned), hipMemcpyDeviceToHost);
        hipMemcpy(centroids, d_centroids, K * DIM * sizeof(float), hipMemcpyDeviceToHost);
        iteration++;
        std::cout << "iteration " << iteration << ": " << (changed[0] ? "changed" : "converged") << std::endl;
        for (unsigned i = 0; i < K; i++)
        {
            std::cout << "cluster " << i << ": ";
            unsigned size = 0;
            for (unsigned j = 0; j < vector_size; j++)
            {
                if (clusters[j] == i)
                {
                    size++;
                }
            }
            std::cout << size << std::endl;
        }
        std::cout << "centroids: " << std::endl;
        for (int i = 0; i < K; i++)
        {
            for (int j = 0; j < DIM; j++)
            {
                std::cout << centroids[i * DIM + j] << " ";
            }
            std::cout << std::endl;
        }
    }

    hipMemcpy(clusters, d_clusters, vector_size * sizeof(unsigned), hipMemcpyDeviceToHost);

    hipFree(d_vectors);
    hipFree(d_centroids);
    hipFree(d_clusters);
    hipFree(d_cluster_sizes);
    hipFree(d_changed);

    clock_gettime(CLOCK_REALTIME, &end_time);

    printf("Total time taken by the GPU part = %lf\n", (double)(end_time.tv_sec - start_time.tv_sec) + (double)(end_time.tv_nsec - start_time.tv_nsec) / 1000000000);

    free(changed);
    free(vectors);
    free(centroids);
    free(clusters);

    return 0;
}
